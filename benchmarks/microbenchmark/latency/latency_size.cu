#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>

#include "function.hpp"

extern "C" size_t function(mignificient::Invocation invocation) {

  int size = reinterpret_cast<const int *>(invocation.payload.data)[0];
  int iters = reinterpret_cast<const int *>(invocation.payload.data)[1];
  printf("Input size %d, iters %d\n", size, iters);

  char *d_ptr;
  char *h_ptr;
  hipMalloc(&d_ptr, sizeof(char) * size);
  hipHostMalloc(&h_ptr, sizeof(char) * size);

  std::vector<long long int> results;
  results.reserve(iters);

  // int iters = 300000;
  hipMemcpy(d_ptr, &h_ptr, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&h_ptr, d_ptr, sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < iters; ++i) {
    auto s = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_ptr, &h_ptr, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&h_ptr, d_ptr, sizeof(float), hipMemcpyDeviceToHost);
    auto e = std::chrono::high_resolution_clock::now();
    auto d =
        std::chrono::duration_cast<std::chrono::nanoseconds>(e - s).count();
    results.push_back(d);
  }

  std::ofstream of{"result.txt", std::ios::out};
  for (int i = 0; i < iters; ++i) {
    of << i << "," << results[i] << std::endl;
  }
  of.close();
  printf("Iters %d\n", iters);
  return 0;
}

