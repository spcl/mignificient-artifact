#include <chrono>
#include <iostream>

#include "function.hpp"

int main() {
  int iters = 300000;
  float x = 1.0;
  float *d_ptr;
  hipMalloc(&d_ptr, sizeof(float));

  // for (int i = 0; i < 300000; ++i) {
  hipMemcpy(d_ptr, &x, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&x, d_ptr, sizeof(float), hipMemcpyDeviceToHost);
  auto s = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < iters; ++i) {
    hipMemcpy(d_ptr, &x, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&x, d_ptr, sizeof(float), hipMemcpyDeviceToHost);
  }
  auto e = std::chrono::high_resolution_clock::now();
  auto d =
      std::chrono::duration_cast<std::chrono::microseconds>(e - s).count() /
      1000000.0;
  printf("%d %.10f\n", iters, d);
  return 0;
}
