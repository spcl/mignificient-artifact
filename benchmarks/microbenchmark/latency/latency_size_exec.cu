#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>

#include "function.hpp"


int main(int argc, char ** argv)
{
  int size = std::stoi(argv[1]);
  int iters = std::stoi(argv[2]);
  printf("Input size %d, iters %d\n", size, iters);

  char *d_ptr;
  char *h_ptr;
  hipMalloc(&d_ptr, sizeof(char) * size);
  hipHostMalloc(&h_ptr, sizeof(char) * size);

  std::vector<long long int> results;
  results.reserve(iters);

  // int iters = 300000;
  hipMemcpy(d_ptr, &h_ptr, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(&h_ptr, d_ptr, sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < iters; ++i) {
    auto s = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_ptr, &h_ptr, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&h_ptr, d_ptr, sizeof(float), hipMemcpyDeviceToHost);
    auto e = std::chrono::high_resolution_clock::now();
    auto d =
        std::chrono::duration_cast<std::chrono::nanoseconds>(e - s).count();
    results.push_back(d);
  }
  printf("Iters %d\n", iters);

  std::ofstream of{"result.txt", std::ios::out};
  for (int i = 0; i < iters; ++i) {
    of << i << "," << results[i] << std::endl;
  }
  of.close();
  return 0;
}
